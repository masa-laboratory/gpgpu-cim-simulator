#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 
 
__global__ void MatMul(int *M,int *N,int *P,int width)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	
	
	float elem1 = 0.0,elem2 = 0.0,value = 0.0;
	for(int i = 0;i < width;i++)
	{
		elem1 = M[y * width + i];//取M矩阵的一行
		elem2 = N[i * width + x];//取N矩阵的一列
		
		value += elem1 * elem2;//求和
	}
	
	P[y * width + x] = value;
}
 
int main()
{
	const int ND = 40;
	int a[ND][ND],b[ND][ND],c[ND][ND];
	int *M,*N,*P;
	
	int width = ND;

	dim3 blockSize(ND,ND);
	
	hipEvent_t start,stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//设备端内存分配
	hipMalloc((void**)&M,ND * ND * sizeof(int));
	hipMalloc((void**)&N,ND * ND * sizeof(int));
	hipMalloc((void**)&P,ND * ND * sizeof(int));
	
	//初始化
	for(int i = 0;i < ND;i++)
	{
		for(int j = 0;j < ND;j++)
		{
			a[i][j] = 2;
			b[i][j] = 3;
		}
	}
	
	int Size = ND * ND;
	//数据拷贝，主机到设备
	hipMemcpy(M,a,Size * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(N,b,Size * sizeof(int),hipMemcpyHostToDevice);
	
	hipEventRecord(start,0);
	MatMul<<<1,blockSize>>>(M,N,P,width);//调用核函数
	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	hipMemcpy(c,P,Size * sizeof(int),hipMemcpyDeviceToHost);
	
	printf("c0 = %d \n",c[0][0]);
	
	//释放设备内存
	hipFree(M);
	hipFree(N);
	hipFree(P);
	
	return 0;
}